#include <hip/hip_runtime.h>

int main() {
    int deviceId;
    hipDeviceProp_t prop;
    hipError_t err;

    err = hipGetDevice(&deviceId);
    if (err != hipSuccess) {
      printf("Failed to get CUDA device: %s\n", hipGetErrorString(err));
      return -1;
    }

    err = hipGetDeviceProperties(&prop, deviceId);
    if (err != hipSuccess) {
      printf("Failed to get device properties: %s\n", hipGetErrorString(err));
      return -1;
    }

    printf("Device Name: %s\n", prop.name);
    printf("CUDA Capability: %d.%d\n", prop.major, prop.minor);

    return 0;
}

