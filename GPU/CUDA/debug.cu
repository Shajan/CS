#include <hip/hip_runtime.h>

int main() {
    // Check device
    int deviceId;
    hipDeviceProp_t prop;
    hipError_t err;

    err = hipGetDevice(&deviceId);
    if (err != hipSuccess) {
      std::cerr << "Failed to get CUDA device: " << hipGetErrorString(err) << std::endl;
      return -1;
    }

    err = hipGetDeviceProperties(&prop, deviceId);
    if (err != hipSuccess) {
      std::cerr << "Failed to get device properties: " << hipGetErrorString(err) << std::endl;
      return -1;
    }

    printf("Device Name: %s\n", prop.name);
    printf("CUDA Capability: %d.%d\n", prop.major, prop.minor);

    return 0;
}

