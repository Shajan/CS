#include <hip/hip_runtime.h>

__global__ void vector_add_kernel(const float* A, const float* B, float* C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N)
        C[idx] = A[idx] + B[idx];

}

void vector_add(const float* A, const float* B, float* C, int N) {
    float *d_A, *d_B, *d_C;

    // Allocate device memory
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));

    // Copy input data from host to device
    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

    // Calculate grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    vector_add_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
